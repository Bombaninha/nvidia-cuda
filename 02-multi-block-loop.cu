
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMBER_OF_BLOCKS 2

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop() {
    int iteration = threadIdx.x + blockIdx.x * blockDim.x;
    printf("%d\n", iteration);
}

int main() {
    /*
    * When refactoring `loop` to launch as a kernel, be sure
    * to use the execution configuration to control how many
    * "iterations" to perform.
    *
    * For this exercise, be sure to use more than 1 block in
    * the execution configuration.
    */
    int N = 10;
    loop<<<NUMBER_OF_BLOCKS, N>>>();
    
    hipDeviceSynchronize();
}
