
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS_PER_BLOCK 1024

__global__ void printSuccessForCorrectExecutionConfiguration() {
    if(threadIdx.x == 1023 && blockIdx.x == 255) {
        printf("Success!\n");
    } 
}

int main() {
    /*
    * Update the execution configuration so that the kernel
    * will print `"Success!"`.
    */
    printSuccessForCorrectExecutionConfiguration<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK>>>();
    
    hipDeviceSynchronize();
}