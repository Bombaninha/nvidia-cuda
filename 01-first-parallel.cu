
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUMBER_OF_BLOCKS 5
#define NUMBER_OF_THREADS_PER_BLOCK 5

/*
 * Refactor firstParallel so that it can run on the GPU.
 */
__global__ void firstParallel() {
  printf("This should be running in parallel.\n");
}

int main() {
    /*
    * Refactor this call to firstParallel to execute in parallel
    * on the GPU.
    */
    firstParallel<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK>>>();

    /*
    * Some code is needed below so that the CPU will wait
    * for the GPU kernels to complete before proceeding.
    */
    hipDeviceSynchronize();
}
